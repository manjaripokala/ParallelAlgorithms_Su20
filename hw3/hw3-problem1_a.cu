#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "Array.c"
#include <time.h>

__global__ void global_reduce_kernel(int * d_out, int * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // do reduction in global mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if ( d_in[myId] < d_in[myId + s]){
                d_in[myId]= d_in[myId + s];
            }
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}

__global__ void shmem_reduce_kernel(int * d_out, const int * d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if ( sdata[tid] < sdata[tid + s]){
                sdata[tid]= sdata[tid + s];
            }
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

void reduce(int * d_out, int * d_intermediate, int * d_in,
            int size, bool usesSharedMemory)
{
    // assumes that size is not greater than maxThreadsPerBlock^2
    // and that size is a multiple of maxThreadsPerBlock
    const int maxThreadsPerBlock = 512;
    int threads = maxThreadsPerBlock;
    int blocks = size / maxThreadsPerBlock;
    if (usesSharedMemory)
    {
        shmem_reduce_kernel<<<blocks, threads, threads * sizeof(int)>>>
                (d_intermediate, d_in);
    }
    else
    {
        global_reduce_kernel<<<blocks, threads>>>
                (d_intermediate, d_in);
    }

    // now we're down to one block left, so reduce it
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    if (usesSharedMemory)
    {
        shmem_reduce_kernel<<<blocks, threads, threads * sizeof(int)>>>
                (d_out, d_intermediate);
    }
    else
    {
        global_reduce_kernel<<<blocks, threads>>>
                (d_out, d_intermediate);
    }
}

int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem,
               (int)devProps.major, (int)devProps.minor,
               (int)devProps.clockRate);
    }
    Array A = initArrayA();
    int * h_in = A.array;
    const int ARRAY_SIZE = A.size;
    const int ARRAY_BYTES = A.size * sizeof(int);

    // generate the input array on the host
    int max = 0;
    printf("array size is %d\n", ARRAY_SIZE);
    clock_t t;
    t = clock();

    for(int i = 0; i < ARRAY_SIZE; i++) {
        if (max <= h_in[i]){
            max = h_in[i];
        }
    }
    t = clock() - t;
    double time_taken = ((double)t)/(CLOCKS_PER_SEC/1000); // calculate the elapsed time
    printf("The host took %f ms to execute\n", time_taken);
    printf("Max at host: %d\n", max);

    // declare GPU memory pointers
    int * d_in, * d_intermediate, * d_out;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
    hipMalloc((void **) &d_out, sizeof(int));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    printf("Running global reduce\n");
    hipEventRecord(start, 0);
    reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, false);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    int h_out;
    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    printf("average time elapsed in ms: %f\n", elapsedTime);
    printf("Max returned by device: %d\n", h_out);

    printf("Running reduce with shared mem\n");
    hipEventRecord(start2, 0);
    reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, true);
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    float elapsedTime2;
    hipEventElapsedTime(&elapsedTime2, start2, stop2);
    int h_out2;
    hipMemcpy(&h_out2, d_out, sizeof(int), hipMemcpyDeviceToHost);
    printf("average time elapsed in ms: %f\n", elapsedTime2);
    printf("Max returned by device: %d\n", h_out2);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);

    return 0;
}
