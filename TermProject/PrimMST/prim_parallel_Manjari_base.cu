
#include <stdio.h> 
#include <queue>
#include <set>
#include <list>
#include <iterator>
#include <algorithm>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 12000
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// Structure to represent a vertex and its distance
struct distNode { 
	int node; 
	int dist; 
  bool operator<(const distNode& rhs) const
    {
        return dist > rhs.dist || (dist == rhs.dist && node > rhs.node);;
    }
}; 

// Structure to represent an edge
struct edge { 
	int from; 
	int to; 
 	int weight;
   	bool operator<(const edge& rhs) const
     {
         return weight > rhs.weight || (weight == rhs.weight && to > rhs.to);
     }
};

// Structure to represent an edge source & destination
struct fromTo { 
	int from; 
	int to; 
   	bool operator<(const fromTo& rhs) const
     {
         return to < rhs.to || (to == rhs.to && from < rhs.from);
     }
};


// Initialize global variables
__device__ __managed__ int parent[ARRAY_SIZE]; // Array to store parent nodes
__device__ __managed__ int dist[ARRAY_SIZE]; // Array to store node distances
__device__ __managed__ bool fixed[ARRAY_SIZE]; // Array to store flags for node traversal
std::vector<bool> nonEmptyIndices; // Array to store non empty indices of vertices

std::priority_queue<distNode> H; //binary heap of (j,dist) initially empty;
__device__ __managed__ int Q[ARRAY_SIZE], R[ARRAY_SIZE]; //set of vertices initially empty;
__device__ __managed__ fromTo T[ARRAY_SIZE*ARRAY_SIZE]; //{ set of edges } initially {};
__device__ __managed__ fromTo mwe[ARRAY_SIZE*ARRAY_SIZE]; //set of edges; minimum weight edges for all vertices
__device__ __managed__ int z_device, Q_index=0, R_index=0, mwe_index=0, T_index=0; //Indices to synchronize between host & device
__device__ __managed__ int edge_cnt=0; //keeps track of #edges

//Arrays to hold all edges of a graph
int allvertex_in[ARRAY_SIZE*ARRAY_SIZE], alledge_in[ARRAY_SIZE*ARRAY_SIZE], allweight_in[ARRAY_SIZE*ARRAY_SIZE];


// class to represent a graph object
class Graph
{
public:
	// construct a vector of vectors of edges to represent an adjacency list
	std::vector<std::vector<edge>> adjList;
	

	// Graph Constructor
	Graph(std::vector<edge> const &edges, int N)
	{
		// resize the vector to hold upto vertex of maximum label value (elements of type vector<edge>)
		//or assign labels to each vertex starting from 0
		adjList.resize(N);
		nonEmptyIndices.resize(N);

		// add edges to the undirected graph
		for (auto &e: edges)
		{
			int from = e.from;
			int to = e.to;
			int weight = e.weight;

			// insert at the end
			adjList[from].push_back(edge{from, to, weight});
			adjList[to].push_back(edge{to, from, weight});

			//flag the non empty indices in adjList
			nonEmptyIndices[from] = true;
			nonEmptyIndices[to] = true;
		}
	}
};


// print adjacency list representation of graph
void printGraph(Graph const &graph)
{
	for (int i = 0; i < graph.adjList.size(); i++)
	{
		// print all neighboring vertices of given vertex
		for (edge v : graph.adjList[i]){
			//printf("( %d, %d, %d )", v.from, v.to, v.weight);
		}
		//printf("\n");
	}
}

//Delete element from array
//template<typename T>
void deleteElement(int arr[], int arr_index, int size) 
{

   if (arr_index < size) 
   { 
     // decrease the size of array and move all elements ahead
     size = size - 1; 
     for (int j=arr_index; j<size; j++) 
        arr[j] = arr[j+1]; 
   }
} 

//Check if an element exists in an array
//template<typename T>
__device__ bool ifExist(int arr[], int val){
		for (int i=0; i<ARRAY_SIZE; i++) {
				if (arr[i] == val)
					return true;
		}
		return false;
}

__device__ bool ifExistMWE(fromTo arr[], fromTo ft){
		for (int i=0; i<edge_cnt; i++) {
				if (arr[i].from == ft.from && arr[i].to == ft.to)
					return true;
		}
		return false;
}


//Function to load edges into kernel pointer arrays
void load_kernelArrays(Graph const &graph) {
	// generate the input array on the host
	//atmost a node can connect to all other nodes
	for (int i = 0; i < graph.adjList.size(); i++) {
		for(edge adj : graph.adjList[i]) {
			allvertex_in[edge_cnt] = adj.from;
			alledge_in[edge_cnt] = adj.to;
			allweight_in[edge_cnt] = adj.weight;
			edge_cnt++;
		}
	}
}

//Identifies all minimum weight edges for all vertices
void initMWE(Graph const &graph) 
{ 
	for (int i = 0; i < graph.adjList.size(); i++) {
		int prevWeight=INT_MAX;
		int min_to, minFrom;
		// Iterate through all the vertices of graph 
		for (auto it=graph.adjList[i].begin(); it!=graph.adjList[i].end(); it++) {
			edge adj = *it;
			// Get the Minimum weight edge for vertex adj.from
			if (adj.weight < prevWeight) { 
				min_to = adj.to;
				minFrom = adj.from;
				prevWeight = adj.weight;
			}
		} 
		mwe[mwe_index] = fromTo{minFrom, min_to};
		mwe_index++;
	}
} 
 
 
//Kernel to process edges in Parallel
__global__ void parallel_processEdge(int *allvertex_devicein, int *alledge_devicein, 
	int *allweight_devicein, int z_device)
{
 
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
	   // int tid  = threadIdx.x;

    //printf("block:%d, myId: %d\n", blockIdx.x, myId); 
	
    // process edges in R
	if (myId < edge_cnt) {
		//printf("myId:%d, allvertex_devicein[myId] :%d\n", myId, allvertex_devicein[myId]);
		if (allvertex_devicein[myId] == z_device)
		{
			//printf("Z found, allvertex_devicein[myId] :%d\n", allvertex_devicein[myId]);
			////printf("block:%d, myId: %d\n", blockIdx.x, myId); 
			int k_device = alledge_devicein[myId];
			//printf("k_device: %d\n", k_device);
			int w_device = allweight_devicein[myId];
			//printf("w_device: %d\n", w_device);

			if (!fixed[k_device]) {
				if (ifExistMWE(mwe, fromTo{z_device, k_device})) {
					//printf("In MWE and not fixed k, z:%d, k:%d\n", z_device, k_device);
					fixed[k_device] = true;
							
					int t = atomicAdd(&T_index, 1);
					T[t] = fromTo{k_device, z_device}; // z is the parent of k
							
					int r = atomicAdd(&R_index, 1);
					R[r] = k_device;
					//printf("R_index in kernel:%d\n", R_index);
				}
				else if (dist[k_device] > w_device) {
					//printf("not minimum edge and not fixed k, z:%d, k:%d\n", z_device, k_device);
					//printf("\n");
					dist[k_device] = w_device;
					parent[k_device] = z_device;

					if (!ifExist(Q, k_device)) {
						int q = atomicAdd(&Q_index, 1);
						Q[q] = k_device;
						//if (Q.find(k_device) == Q.end()) {
						//	Q.insert(k_device);
					}
				}
			}
			__syncthreads();        // make sure all updates are finished
		}
	}
}

//Kernel Setup
void kernel_setup(Graph const &graph, int z_device){
	
	int threads = 512;
	int blocks = ceil(float(edge_cnt) / float(threads));

	const int ARRAY_BYTES = ARRAY_SIZE * ARRAY_SIZE * sizeof(int);
	//printf("array bytes:%f\n", ARRAY_BYTES);

	// declare GPU memory pointers
    int * allvertex_devicein, * alledge_devicein, * allweight_devicein;

    // allocate GPU memory
    hipMalloc((void **) &allvertex_devicein, ARRAY_BYTES);
	hipMalloc((void **) &alledge_devicein, ARRAY_BYTES);
	hipMalloc((void **) &allweight_devicein, ARRAY_BYTES);

	// transfer the input array to the GPU
	hipMemcpy(allvertex_devicein, allvertex_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	gpuErrchk( hipMemcpy(alledge_devicein, alledge_in, ARRAY_BYTES, hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(allweight_devicein, allweight_in, ARRAY_BYTES, hipMemcpyHostToDevice) );


	//printf("Running global reduce\n");
	
	parallel_processEdge<<<blocks, threads>>>
		(allvertex_devicein, alledge_devicein, allweight_devicein, z_device);

	gpuErrchk( hipPeekAtLastError() );
  	gpuErrchk( hipDeviceSynchronize() );

	// free GPU memory allocation
	hipFree(allvertex_devicein);
	hipFree(alledge_devicein);
	hipFree(allweight_devicein);
};
 

// Function to print the constructed MST 
void printMST(std::set<fromTo> T) 
{ 
	std::set<fromTo>::iterator it; //set iterator
	for (it=T.begin(); it!=T.end(); it++) {
		fromTo e = *it; 
		printf("%d - %d\n", e.from, e.to); 
  }
} 

// The main function that constructs Minimum Spanning Tree (MST) 
// using Prim's Parallel algorithm given in chapter 7
fromTo* primMST(Graph const &graph, int N, int source) 
{ 
	std::set<int>::iterator it; //set iterator 

	// Initialize and assign dist value of 
	// all vertices to 0 and source to infinite 
	for(int i = 0; i < N; i ++) {
		parent[i] = -1;
		dist[i] = INT_MAX;
		fixed[i] = false;
	}

	// Make distance value of source vertex as 0 so it is extracted first 
	dist[source] = 0; 
	H.push(distNode{source, dist[0]});

	initMWE(graph); //initialize minimum weight edges of given graph;

	// Loop for |V| - 1 iterations
	//while (!H.empty()) { 
	for (int i = 0; i < graph.adjList.size(); i++) {
		// Extract the vertex with minimum dist value 
		distNode d = H.top();
		H.pop();
		int j = d.node; //pop the minimum distance vertex
		//printf("Pop min distance node:%d\n", j);
		if (!fixed[j]) {
			R[R_index] = j;
			R_index++;
			fixed[j] = true;
			if (parent[j] != -1) {
				T[T_index] = fromTo{j, parent[j]};
				T_index++;
			}

			//printf("R_index: %d\n", R_index);
			while (R_index != 0){
				// call processEdge for all neighbors of vertex in R 
				//printf("R_index: %d\n", R_index);
				z_device = R[0];
				//printf("Z before kernel:%d\n", z_device);
				deleteElement(R, 0, ARRAY_SIZE);
				R_index--;
				//call kernel setup
				kernel_setup(graph, z_device);
			}	
			
			while (Q_index != 0) {
				for (int i = 0; i < Q_index; i++) {
					int z = Q[i];
					//printf("z in Q:%d\n", z);
					deleteElement(Q, i, ARRAY_SIZE);
					Q_index--;
					if (!fixed[z]) {
						H.push(distNode{z, dist[z]});
					}
				}
			}
		}
	}
	if (T_index == graph.adjList.size() -1) {
		return T;
	} else 
		return new fromTo[ARRAY_SIZE]; // return empty tree

} 

// Driver program to call Prim
int main() 
{ 
	printf("2010_Alaskan.txt\n");
	// vector of graph edges
	std::vector<edge> edges;

	// START 

	// STOP

	// construct graph
	Graph graph(edges, ARRAY_SIZE);
	load_kernelArrays(graph);

	// print adjacency list representation of graph
	printGraph(graph);

	//Source vertex as first non empty vertex in adjacency List
	//Or modify this to take from input file
	int source;
	for(int i = 0; i<nonEmptyIndices.size(); i++) {
			if (nonEmptyIndices[i]) {
				source = i;
			break;
		}
	}	
	
	//printf("source:%d\n", source);
	
  	//printf("Before Prim\n");

	hipEvent_t start, stop;
    	hipEventCreate(&start);
        hipEventCreate(&stop);

        //printf("Running global reduce\n");
    	hipEventRecord(start, 0);

	primMST(graph, ARRAY_SIZE, source);

	 hipEventRecord(stop, 0);

        hipEventSynchronize(stop);
    	float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Parallel Elpased Time in ms:%f\n", elapsedTime);

	
  	//printf("After Prim\n");

	//printf("T size:%d\n", T_index);
	//printf("MST in iterator\n");
	for (int i =0; i<T_index; i++) {
		fromTo e = T[i]; 
		printf("%d - %d\n", e.from, e.to); 
	}

	
	return 0; 
} 

//Reference: https://www.geeksforgeeks.org/prims-mst-for-adjacency-list-representation-greedy-algo-6/
// https://www.techiedelight.com/graph-implementation-using-stl/

