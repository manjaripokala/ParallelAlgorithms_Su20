// C / C++ program for Prim's MST for adjacency list representation of graph 



#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <queue>
#include <set>
#include <list>
#include <iterator>
#include <algorithm>


#define ARRAY_SIZE 15
//#define ARRAY_BYTES (15 * 15 * sizeof(int));
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// Structure to represent a vertex and its distance
struct distNode { 
	int node; 
	int dist; 
  bool operator<(const distNode& rhs) const
    {
        return dist > rhs.dist || (dist == rhs.dist && node > rhs.node);;
    }
}; 

// Structure to represent an edge
struct edge { 
	int from; 
	int to; 
 	int weight;
   	bool operator<(const edge& rhs) const
     {
         return weight > rhs.weight || (weight == rhs.weight && to > rhs.to);
     }
};

// Structure to represent a edge source & destination
struct fromTo { 
	int from; 
	int to; 
   	bool operator<(const fromTo& rhs) const
     {
         return to < rhs.to || (to == rhs.to && from < rhs.from);
     }
};


// Initialize global variables
__device__ __managed__ int parent[ARRAY_SIZE]; // Vector to store parent nodes
__device__ __managed__ int dist[ARRAY_SIZE]; // dist values used to pick minimum weight edge in cut 
__device__ __managed__ bool fixed[ARRAY_SIZE]; // Vector to store flags for node traversal
std::vector<bool> nonEmptyIndices; // Vector to store non empty indices of vertices

std::priority_queue<distNode> H; //binary heap of (j,dist) initially empty;
__device__ __managed__ int Q[ARRAY_SIZE], R[ARRAY_SIZE]; //set of vertices initially empty;
//__device__ __managed__ std::set<int> R, Q;
__device__ __managed__ fromTo T[ARRAY_SIZE*ARRAY_SIZE]; //{ set of edges } initially {};
__device__ __managed__ fromTo mwe[ARRAY_SIZE*ARRAY_SIZE]; //set of edges; minimum weight edges for all vertices
__device__ __managed__ int z_device, Q_index=0, R_index=0, mwe_index=0, T_index=0; //Indices to synchronize between host & device
__device__ __managed__ int edge_cnt=0; //keeps track of #edges


// class to represent a graph object
class Graph
{
public:
	// construct a vector of vectors of edges to represent an adjacency list
	std::vector<std::vector<edge>> adjList;
	
	//Graph Vectors
// std::vector<int> vertices; //Vector to hold nodes of graph
// std::vector<int> edges; //Vector to hold edges of graph
// std::vector<int> weights; //Vector to hold weights of graph

	// Graph Constructor
	Graph(std::vector<edge> const &edges, int N)
	{
		// resize the vector to hold upto vertex of maximum label value (elements of type vector<edge>)
		adjList.resize(N);
		// edges.resize(N);
		// weights.resize(N);
		nonEmptyIndices.resize(N);

		// add edges to the undirected graph
		for (auto &e: edges)
		{
			int from = e.from;
			int to = e.to;
			int weight = e.weight;

			// insert at the end
			adjList[from].push_back(edge{from, to, weight});
			adjList[to].push_back(edge{to, from, weight});

			//flag the non empty indices in adjList
			nonEmptyIndices[from] = true;
			nonEmptyIndices[to] = true;
		}
	}
};


// // A utility function to add an edge in an 
// // undirected graph. 
// void addEdge(std::vector<edge> adj[], int u, int v) 
// { 
//     adj[u].push_back(edge{u,v}); 
//     adj[v].push_back(edge{v,u}); 
// } 

// print adjacency list representation of graph
void printGraph(Graph const &graph)
{
	for (int i = 0; i < graph.adjList.size(); i++)
	{
		// print all neighboring vertices of given vertex
		for (edge v : graph.adjList[i]){
			printf("( %d, %d, %d )", v.from, v.to, v.weight);
		}
		printf("\n");
	}
}

//Delete element from array
template<typename T>
void deleteElement(T arr[], int arr_index, int size) 
{

   if (arr_index < size) 
   { 
     // reduce size of array and move all 
     // elements on space ahead 
     size = size - 1; 
     for (int j=arr_index; j<size; j++) 
        arr[j] = arr[j+1]; 
   }
} 

template<typename T>
__device__ bool ifExist(T arr[], T val){
		for (int i=0; i<ARRAY_SIZE; i++) {
				if (arr[i] == val)
					return true;
		}
		return false;
}

__device__ bool ifExistMWE(fromTo arr[], fromTo ft){
		for (int i=0; i<edge_cnt; i++) {
				if (arr[i].from == ft.from && arr[i].to == ft.to)
					return true;
		}
		return false;
}

template<typename T>
int getIndex(T arr[]){
		printf("sizeof(arr):%d\n", sizeof(arr));
		printf("sizeof(T):%d\n", sizeof(T));
		printf("sizeof(arr)/sizeof(arr[0]):%d\n", sizeof(arr)/sizeof(arr[0]));
		return sizeof(arr)/sizeof(T);
}

//Identifies all minimum weight edges for all vertices
void initMWE(Graph const &graph) 
{ 
	for (int i = 0; i < graph.adjList.size(); i++) {
		// Extract the vertex with minimum dist value 
		int prevWeight=INT_MAX;
		int min_to, minFrom;
		// Iterate through all the vertices of graph 
		//for (edge adj : graph.adjList[i]) {
		for (auto it=graph.adjList[i].begin(); it!=graph.adjList[i].end(); it++) {
			edge adj = *it;
			// Get the Minimum weight edge for vertex adj.from
			if (adj.weight < prevWeight) { 
				min_to = adj.to;
				minFrom = adj.from;
				prevWeight = adj.weight;
			}
		} 
		//printf("minFrom:%d, min_to:%d\n", minFrom, min_to);
		mwe[mwe_index] = fromTo{minFrom, min_to};
		mwe_index++;
	}
} 

// Get Weight for an edge
int getWeight(Graph const &graph, int u, int v) {
	int weight;
	// Iterate through all adjacent vertices of u and extract weight of u to v edge
	for (edge adj : graph.adjList[u]) {
		// Get the Minimum weight edge for vertex v.from
		if (adj.to == v) { 			
			weight = adj.weight;
		}
	}
	return weight;
}


// Process Edge in Parallel
//__device__ void processEdge1(Graph const &graph, int z, int k)
// __device__ void processEdge1(int *allvertex_devicein, int *alledge_devicein, 
// 	int *allweight_devicein, int z_device, int k_device) 
// { 
// 	int weight;
// 	for(int i=0; i<edge_cnt; i++) {
// 		if (allvertex_devicein[i] == z_device && alledge_devicein[i] == k_device) {
// 			weight = allweight_devicein[i];
// 		}
// 	}
// 	if (mwe.find(fromTo{z_device, k_device}) != mwe.end()) {
// 		fixed[k_device] = true;
// 		T.insert(fromTo{k_device, z_device}); // z is the parent of k
// 		R.insert(k_device);
// 	}
// 	else if (dist[k_device] > weight) {
// 		dist[k_device] = weight;
// 		parent[k_device] = z_device;
// 		if (Q.find(k_device) == Q.end()) {
// 			Q.insert(k_device);
// 		}
// 	}
// }
 
 
//Kernel to process edges in Parallel
__global__ void parallel_processEdge(int *allvertex_devicein, int *alledge_devicein, 
	int *allweight_devicein, int z_device)
//, int R_index, int Q_index, int T_index)
{
 
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
	   // int tid  = threadIdx.x;

    //printf("block:%d, myId: %d\n", blockIdx.x, myId); 
	
    // stride is the total number of threads in the grid
    // Using stride increases the performance and benefits with scalability & thread reusage
    //int stride = blockDim.x * gridDim.x;
    
    // do counts in global mem
		//for(int i = 0; i<edge_cnt; i++){
		if (myId < edge_cnt) {
			if (allvertex_devicein[myId] == z_device)
			{
					printf("allvertex_devicein[myId] :%d\n", allvertex_devicein[myId]);
				int k_device = alledge_devicein[myId];
				printf("k_device: %d\n", k_device);
				int w_device = allweight_devicein[myId];
				printf("w_device: %d\n", w_device);

					if (!fixed[k_device]) {
						if (ifExistMWE(mwe, fromTo{z_device, k_device})) {
							printf("In MWE and not fixed k, z:%d, k:%d\n", z_device, k_device);
							fixed[k_device] = true;
							
							int t = atomicAdd(&T_index, 1);
							T[t] = fromTo{k_device, z_device}; // z is the parent of k
							
							int r = atomicAdd(&R_index, 1);
							R[r] = k_device;
							printf("R_index in kernel:%d\n", R_index);
						}
						else if (dist[k_device] > w_device) {
								printf("not minimum edge and not fixed k, z:%d, k:%d\n", z_device, k_device);
								printf("\n");
							dist[k_device] = w_device;
							parent[k_device] = z_device;

							if (!ifExist(Q, k_device)) {
								int q = atomicAdd(&Q_index, 1);
								Q[q] = k_device;
							//if (Q.find(k_device) == Q.end()) {
							//	Q.insert(k_device);
							}
						}
						//processEdge1(allvertex_devicein, alledge_devicein, allweight_devicein, z_device, k_device);
			}
					__syncthreads();        // make sure all adds at one stage are done!
			}
		}
}

//Kernel Setup
void kernel_setup(Graph const &graph, int z_device){
	
	int threads = 8;
	    int blocks = ceil(float(ARRAY_SIZE) / float(threads));

	
	const int ARRAY_BYTES = ARRAY_SIZE * ARRAY_SIZE * sizeof(int);
	printf("array bytes:%f\n", ARRAY_BYTES);

	//declare GPU pointers
	// generate the input array on the host
	//atmost a node can connect to all other nodes
	int allvertex_in[ARRAY_SIZE*ARRAY_SIZE], alledge_in[ARRAY_SIZE*ARRAY_SIZE], allweight_in[ARRAY_SIZE*ARRAY_SIZE];
	
	
	for (int i = 0; i < graph.adjList.size(); i++) {
		for(edge adj : graph.adjList[i]) {
			allvertex_in[edge_cnt] = adj.from;
			alledge_in[edge_cnt] = adj.to;
			allweight_in[edge_cnt] = adj.weight;
			edge_cnt++;
		}
	}

	//for (int i = 0; i <edge_cnt; i++) {
	//	printf("allvertex_in:%d, alledge_in:%d, allweight_in:%d\n", allvertex_in[i], alledge_in[i], allweight_in[i]);
	//}

	// declare GPU memory pointers
    int * allvertex_devicein, * alledge_devicein, * allweight_devicein;

    // allocate GPU memory
    hipMalloc((void **) &allvertex_devicein, ARRAY_BYTES);
	hipMalloc((void **) &alledge_devicein, ARRAY_BYTES);
	hipMalloc((void **) &allweight_devicein, ARRAY_BYTES);

	// transfer the input array to the GPU
	hipMemcpy(allvertex_devicein, allvertex_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	gpuErrchk( hipMemcpy(alledge_devicein, alledge_in, ARRAY_BYTES, hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(allweight_devicein, allweight_in, ARRAY_BYTES, hipMemcpyHostToDevice) );


	hipEvent_t start, stop;
    hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("Running global reduce\n");
    hipEventRecord(start, 0);
	
	parallel_processEdge<<<blocks, threads>>>
						(allvertex_devicein, alledge_devicein, allweight_devicein, z_device);
						//, R_index, Q_index, T_index);
	gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

	// Increment index pointers
	//R_index = R_index + threads + 1;
	//Q_index = Q_index + threads + 1;
	//T_index = T_index + threads + 1;

	gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

		// free GPU memory allocation
	hipFree(allvertex_devicein);
	hipFree(alledge_devicein);
	hipFree(allweight_devicein);
};
 

// A utility function used to print the constructed MST 
void printMST(std::set<fromTo> T) 
{ 
	std::set<fromTo>::iterator it; //set iterator
	for (it=T.begin(); it!=T.end(); it++) {
		fromTo e = *it; 
		printf("%d - %d\n", e.from, e.to); 
  }
} 

// The main function that constructs Minimum Spanning Tree (MST) 
// using Prim's Parallel algorithm 
fromTo* primMST(Graph const &graph, int N, int source) 
{ 
	std::set<int>::iterator it; //set iterator 
	
	printf("In PRIM\n");
	fflush( stdout );

	// Initialize min heap with all vertices. dist value of 
	// all vertices (except 0th vertex) is initially infinite 
 
	for(int i = 0; i < N; i ++) {
		parent[i] = -1;
		dist[i] = INT_MAX;
		fixed[i] = false;
	}

	// Make distance value of source vertex as 0 so it is extracted first 
	dist[source] = 0; 
	H.push(distNode{source, dist[0]});

	initMWE(graph); //initialize minimum weight edges of given graph;

	// Loop for |V| - 1 iterations
	//while (!H.empty()) { 
	for (int i = 0; i < graph.adjList.size(); i++) {
		// Extract the vertex with minimum dist value 
		distNode d = H.top();
		H.pop();
		int j = d.node; //pop the minimum distance vertex
		printf("Pop min distance node:%d\n", j);
		if (!fixed[j]) {
			R[R_index] = j;
			R_index++;
			fixed[j] = true;
			if (parent[j] != -1) {
				T[T_index] = fromTo{j, parent[j]};
				T_index++;
			}

			
			while (R_index != 0){
					// call processEdge for all neighbors of vertex in R 
					printf("R_index: %d\n", R_index);
					z_device = R[0];
					deleteElement(R, 0, ARRAY_SIZE);
					R_index--;
					//allocate pointers copy required inputs to device
					//int *z_device;
					//cudaMemcpy(fixed_device, fixed, ARRAY_BYTES, cudaMemcpyHostToDevice);
					//cudaMemcpy(z_device, z, sizeof(int), cudaMemcpyHostToDevice);
					//call kernel setup
					
					kernel_setup(graph, z_device);
					

					//copy back updated values from device
					//cudaMemcpy(&fixed, fixed_device, ARRAY_BYTES, cudaMemcpyDeviceToHost);

					// for (edge adj : graph.adjList[z]) {
					// 	int k = adj.to; 
					// 	if (!fixed[k]) {
					// 		processEdge1(graph, z, k);							 
					// }
				//}
			}	
			
			while (Q_index != 0) {
				for (int i = 0; i < Q_index; i++) {
					int z = Q[i];
					printf("z in Q:%d\n", z);
					deleteElement(Q, i, ARRAY_SIZE);
					Q_index--;
					if (!fixed[z]) {
						H.push(distNode{z, dist[z]});
					}
				}
			}
		}
	}
	if (T_index == graph.adjList.size() -1) {
		return T;
	} else 
		return new fromTo[ARRAY_SIZE]; // return empty tree

} 

// Driver program to test above functions 
int main() 
{ 
	// vector of graph edges as per above diagram.
	// Please note that initialization vector in below format will
	// work fine in C++11, C++14, C++17 but will fail in C++98.
	std::vector<edge> edges;
	edges.push_back(edge{4,5,4});
	edges.push_back(edge{4,11,8});
	edges.push_back(edge{5,6,8});
	edges.push_back(edge{5,11,11});
	edges.push_back(edge{6,7,7});
	edges.push_back(edge{6,12,2});
	edges.push_back(edge{6,9,4});
	edges.push_back(edge{7,8,9});
	edges.push_back(edge{7,9,14});
	edges.push_back(edge{8,9,10});
	edges.push_back(edge{9,10,2});
	edges.push_back(edge{10,11,1});
	edges.push_back(edge{10,12,6});
	edges.push_back(edge{11,12,7});
		
	// Maxmum label value of vertices in the given graph, assume 1000
	//int N = 15;
	
	

	//create vertex, edge, weight arrays on host
	//const int ARRAY_SIZE = 15;
    //const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int) * ARRAY_SIZE;

	// construct graph
	Graph graph(edges, ARRAY_SIZE);

	// print adjacency list representation of graph
	printGraph(graph);

	//Source vertex as first non empty vertex in adjacency List
	int source;
	for(int i = 0; i<nonEmptyIndices.size(); i++) {
			if (nonEmptyIndices[i]) {
				source = i;
			break;
		}
	}

    // // generate the input array on the host
    // int allvertex_in[ARRAY_SIZE], alledge_in[ARRAY_SIZE], allweight_in[ARRAY_SIZE];
	// int i = 0;
	//uncomment this while reading from input file
    //for(int i = 0; i < ARRAY_SIZE; i++) {
	// for (auto &e: edges) {
    //     // generate input array of vertices, edges, weights
	// 	allvertex_in[i] = e.from;
	// 	alledge_in[i] = e.to;
	// 	allweight_in[i] = e.weight;

	// 	allvertex_in[i+] = e.from;
	// 	alledge_in[e.from] = e.to;
	// 	allweight_in[e.from] = e.weight;
    //     i++;
    // }
    // printf("count at host: %d\n", count);

	// declare GPU memory pointers
	//std::vector<std::vector<edge>> * adjList_devicein;
    //int * d_in, * d_intermediate, * d_out;

	
	
	printf("source:%d\n", source);
	
  printf("Before Prim\n");
  fflush( stdout );

	primMST(graph, ARRAY_SIZE, source);
	
  printf("After Prim\n");
  fflush( stdout );

	printf("T size:%d\n", T_index);
	printf("MST in iterator\n");
	for (int i =0; i<T_index; i++) {
		fromTo e = T[i]; 
		printf("%d - %d\n", e.from, e.to); 
	}

	
	return 0; 
} 

//Reference: https://www.geeksforgeeks.org/prims-mst-for-adjacency-list-representation-greedy-algo-6/
// https://www.techiedelight.com/graph-implementation-using-stl/
