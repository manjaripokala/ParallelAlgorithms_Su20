
#include <stdio.h> 
#include <queue>
#include <set>
#include <list>
#include <iterator>
#include <algorithm>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 30
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// Structure to represent a vertex and its distance
struct distNode { 
	int node; 
	int dist; 
  bool operator<(const distNode& rhs) const
    {
        return dist > rhs.dist || (dist == rhs.dist && node > rhs.node);;
    }
}; 

// Structure to represent an edge
struct edge { 
	int from; 
	int to; 
 	int weight;
   	bool operator<(const edge& rhs) const
     {
         return weight > rhs.weight || (weight == rhs.weight && to > rhs.to);
     }
};

// Structure to represent an edge source & destination
struct fromTo { 
	int from; 
	int to; 
   	bool operator<(const fromTo& rhs) const
     {
         return to < rhs.to || (to == rhs.to && from < rhs.from);
     }
};


// Initialize global variables
__device__ __managed__ int parent[ARRAY_SIZE]; // Array to store parent nodes
__device__ __managed__ int dist[ARRAY_SIZE]; // Array to store node distances
__device__ __managed__ bool fixed[ARRAY_SIZE]; // Array to store flags for node traversal
std::vector<bool> nonEmptyIndices; // Array to store non empty indices of vertices

std::priority_queue<distNode> H; //binary heap of (j,dist) initially empty;
__device__ __managed__ int Q[ARRAY_SIZE], R[ARRAY_SIZE]; //set of vertices initially empty;
__device__ __managed__ fromTo T[ARRAY_SIZE*ARRAY_SIZE]; //{ set of edges } initially {};
__device__ __managed__ fromTo mwe[ARRAY_SIZE*ARRAY_SIZE]; //set of edges; minimum weight edges for all vertices
__device__ __managed__ int z_device, Q_index=0, R_index=0, mwe_index=0, T_index=0; //Indices to synchronize between host & device
__device__ __managed__ int edge_cnt=0; //keeps track of #edges

//Arrays to hold all edges of a graph
int allvertex_in[ARRAY_SIZE*ARRAY_SIZE], alledge_in[ARRAY_SIZE*ARRAY_SIZE], allweight_in[ARRAY_SIZE*ARRAY_SIZE];


// class to represent a graph object
class Graph
{
public:
	// construct a vector of vectors of edges to represent an adjacency list
	std::vector<std::vector<edge>> adjList;
	

	// Graph Constructor
	Graph(std::vector<edge> const &edges, int N)
	{
		// resize the vector to hold upto vertex of maximum label value (elements of type vector<edge>)
		//or assign labels to each vertex starting from 0
		adjList.resize(N);
		nonEmptyIndices.resize(N);

		// add edges to the undirected graph
		for (auto &e: edges)
		{
			int from = e.from;
			int to = e.to;
			int weight = e.weight;

			// insert at the end
			adjList[from].push_back(edge{from, to, weight});
			adjList[to].push_back(edge{to, from, weight});

			//flag the non empty indices in adjList
			nonEmptyIndices[from] = true;
			nonEmptyIndices[to] = true;
		}
	}
};


// print adjacency list representation of graph
void printGraph(Graph const &graph)
{
	printf("Input Graph\n");
	for (int i = 0; i < graph.adjList.size(); i++)
	{
		// print all neighboring vertices of given vertex
		for (edge v : graph.adjList[i]){
			printf("( %d, %d, %d )", v.from, v.to, v.weight);
		}
		//printf("\n");
	}
}

//Delete element from array
//template<typename T>
void deleteElement(int arr[], int arr_index, int size) 
{

   if (arr_index < size) 
   { 
     // decrease the size of array and move all elements ahead
     size = size - 1; 
     for (int j=arr_index; j<size; j++) 
        arr[j] = arr[j+1]; 
   }
} 

//Check if an element exists in an array
//template<typename T>
__device__ bool ifExist(int arr[], int val){
		for (int i=0; i<ARRAY_SIZE; i++) {
				if (arr[i] == val)
					return true;
		}
		return false;
}

__device__ bool ifExistMWE(fromTo arr[], fromTo ft){
		for (int i=0; i<edge_cnt; i++) {
				if (arr[i].from == ft.from && arr[i].to == ft.to)
					return true;
		}
		return false;
}


//Function to load edges into kernel pointer arrays
void load_kernelArrays(Graph const &graph) {
	// generate the input array on the host
	//atmost a node can connect to all other nodes
	for (int i = 0; i < graph.adjList.size(); i++) {
		for(edge adj : graph.adjList[i]) {
			allvertex_in[edge_cnt] = adj.from;
			alledge_in[edge_cnt] = adj.to;
			allweight_in[edge_cnt] = adj.weight;
			edge_cnt++;
		}
	}
}

//Identifies all minimum weight edges for all vertices
void initMWE(Graph const &graph) 
{ 
	for (int i = 0; i < graph.adjList.size(); i++) {
		int prevWeight=INT_MAX;
		int min_to, minFrom;
		// Iterate through all the vertices of graph 
		for (auto it=graph.adjList[i].begin(); it!=graph.adjList[i].end(); it++) {
			edge adj = *it;
			// Get the Minimum weight edge for vertex adj.from
			if (adj.weight < prevWeight) { 
				min_to = adj.to;
				minFrom = adj.from;
				prevWeight = adj.weight;
			}
		} 
		mwe[mwe_index] = fromTo{minFrom, min_to};
		mwe_index++;
	}
} 
 
 
//Kernel to process edges in Parallel
__global__ void parallel_processEdge(int *allvertex_devicein, int *alledge_devicein, 
	int *allweight_devicein, int z_device)
{
 
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
	   // int tid  = threadIdx.x;

    //printf("block:%d, myId: %d\n", blockIdx.x, myId); 
	
    // process edges in R
	if (myId < edge_cnt) {
		//printf("myId:%d, allvertex_devicein[myId] :%d\n", myId, allvertex_devicein[myId]);
		if (allvertex_devicein[myId] == z_device)
		{
			//printf("Z found, allvertex_devicein[myId] :%d\n", allvertex_devicein[myId]);
			////printf("block:%d, myId: %d\n", blockIdx.x, myId); 
			printf("Thread %d looking for the Edge to be processed\n", threadIdx.x);
			int k_device = alledge_devicein[myId];
			//printf("k_device: %d\n", k_device);
			int w_device = allweight_devicein[myId];
			//printf("w_device: %d\n", w_device);
			printf("Edge {%d, %d, %d} found at myID:%d\n", z_device, k_device, w_device);

			if (!fixed[k_device]) {
				if (ifExistMWE(mwe, fromTo{z_device, k_device})) {
					//printf("In MWE and not fixed k, z:%d, k:%d\n", z_device, k_device);
					fixed[k_device] = true;
							
					int t = atomicAdd(&T_index, 1);
					T[t] = fromTo{k_device, z_device}; // z is the parent of k
							
					int r = atomicAdd(&R_index, 1);
					R[r] = k_device;
					printf("Destination node is not fixed & also a minimum edge for Z:%d\n", z_device);
					printf("Adding k:%d to Tree & R for processing\n", k_device);
					//printf("R_index in kernel:%d\n", R_index);
				}
				else if (dist[k_device] > w_device) {
					//printf("not minimum edge and not fixed k, z:%d, k:%d\n", z_device, k_device);
					//printf("\n");
					printf("Destination node is not fixed & NOT a minimum weight edge\n");
					printf("Adding k:%d to Q for inserting into Heap\n", k_device);
					dist[k_device] = w_device;
					parent[k_device] = z_device;

					if (!ifExist(Q, k_device)) {
						int q = atomicAdd(&Q_index, 1);
						Q[q] = k_device;
						//if (Q.find(k_device) == Q.end()) {
						//	Q.insert(k_device);
					}
				}
			}
			__syncthreads();        // make sure all updates are finished
		}
	}
}

//Kernel Setup
void kernel_setup(Graph const &graph, int z_device){
	
	int threads = 512;
	int blocks = ceil(float(edge_cnt) / float(threads));

	const int ARRAY_BYTES = ARRAY_SIZE * ARRAY_SIZE * sizeof(int);
	//printf("array bytes:%f\n", ARRAY_BYTES);

	// declare GPU memory pointers
    int * allvertex_devicein, * alledge_devicein, * allweight_devicein;

    // allocate GPU memory
    hipMalloc((void **) &allvertex_devicein, ARRAY_BYTES);
	hipMalloc((void **) &alledge_devicein, ARRAY_BYTES);
	hipMalloc((void **) &allweight_devicein, ARRAY_BYTES);

	// transfer the input array to the GPU
	hipMemcpy(allvertex_devicein, allvertex_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	gpuErrchk( hipMemcpy(alledge_devicein, alledge_in, ARRAY_BYTES, hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(allweight_devicein, allweight_in, ARRAY_BYTES, hipMemcpyHostToDevice) );


	//printf("Running global reduce\n");
	
	parallel_processEdge<<<blocks, threads>>>
		(allvertex_devicein, alledge_devicein, allweight_devicein, z_device);

	gpuErrchk( hipPeekAtLastError() );
  	gpuErrchk( hipDeviceSynchronize() );

	// free GPU memory allocation
	hipFree(allvertex_devicein);
	hipFree(alledge_devicein);
	hipFree(allweight_devicein);
};
 

// Function to print the constructed MST 
void printMST(std::set<fromTo> T) 
{ 
	std::set<fromTo>::iterator it; //set iterator
	for (it=T.begin(); it!=T.end(); it++) {
		fromTo e = *it; 
		printf("%d - %d\n", e.from, e.to); 
  }
} 

// The main function that constructs Minimum Spanning Tree (MST) 
// using Prim's Parallel algorithm given in chapter 7
fromTo* primMST(Graph const &graph, int N, int source) 
{ 
	std::set<int>::iterator it; //set iterator 

	// Initialize and assign dist value of 
	// all vertices to 0 and source to infinite 
	for(int i = 0; i < N; i ++) {
		parent[i] = -1;
		dist[i] = INT_MAX;
		fixed[i] = false;
	}

	// Make distance value of source vertex as 0 so it is extracted first 
	dist[source] = 0; 
	H.push(distNode{source, dist[0]});

	initMWE(graph); //initialize minimum weight edges of given graph;

	// Loop for |V| - 1 iterations
	//while (!H.empty()) { 
	for (int i = 0; i < graph.adjList.size(); i++) {
		// Extract the vertex with minimum dist value 
		distNode d = H.top();
		H.pop();
		int j = d.node; //pop the minimum distance vertex
		printf("Popped minimum distance node:%d\n", j);
		if (!fixed[j]) {
			printf("Popped node is not fixed adding it to R\n");
			R[R_index] = j;
			R_index++;
			fixed[j] = true;
			if (parent[j] != -1) {
				T[T_index] = fromTo{j, parent[j]};
				T_index++;
			}

			//printf("R_index: %d\n", R_index);
			while (R_index != 0){
				// call processEdge for all neighbors of vertex in R 
				//printf("R_index: %d\n", R_index);
				z_device = R[0];
				//printf("Z before kernel:%d\n", z_device);
				deleteElement(R, 0, ARRAY_SIZE);
				R_index--;
				//call kernel setup
				printf("Calling kernel for processing edges of elements in R in parallel\n");
				kernel_setup(graph, z_device);
			}	
			
			while (Q_index != 0) {
				//for (int i = 0; i < Q_index; i++) {
					printf("Adding all elements from Q to Heap H\n");
					int z = Q[0];
					//printf("z in Q:%d\n", z);
					deleteElement(Q, 0, ARRAY_SIZE);
					Q_index--;
					if (!fixed[z]) {
						H.push(distNode{z, dist[z]});
					}
				//}
			}
		}
	}
	if (T_index == graph.adjList.size() -1) {
		return T;
	} else 
		return new fromTo[ARRAY_SIZE]; // return empty tree

} 

// Driver program to call Prim
int main() 
{ 
	printf("2010_Alaskan.txt\n");
	// vector of graph edges
    std::vector<edge> edges;
    
    edges.push_back(edge{0, 1, 866});
    edges.push_back(edge{0, 2, 187});
    edges.push_back(edge{0, 3, 399});

    edges.push_back(edge{1, 5, 605});
    edges.push_back(edge{1, 10, 1720});
    edges.push_back(edge{1, 11, 888});
    edges.push_back(edge{1, 12, 409});

    edges.push_back(edge{2, 1, 739});
    edges.push_back(edge{2, 3, 213});
    edges.push_back(edge{2, 4, 541});
    edges.push_back(edge{2, 5, 759});
    edges.push_back(edge{2, 6, 1416});
    edges.push_back(edge{2, 7, 1391});
    edges.push_back(edge{2, 8, 2474});
    edges.push_back(edge{2, 9, 2586});
    edges.push_back(edge{2, 10, 2421});
    edges.push_back(edge{2, 11, 1625});
    edges.push_back(edge{2, 12, 765});

    edges.push_back(edge{3, 4, 330});
    edges.push_back(edge{3, 5, 547});
    edges.push_back(edge{3, 12, 561});

    edges.push_back(edge{4, 5, 226});
    edges.push_back(edge{4, 6, 912});

    edges.push_back(edge{5, 6, 689});
    edges.push_back(edge{5, 7, 731});
    edges.push_back(edge{5, 11, 1199});
    edges.push_back(edge{5, 12, 213});

    edges.push_back(edge{6, 7, 224});
    edges.push_back(edge{6, 8, 1378});

    edges.push_back(edge{7, 8, 1234});
    edges.push_back(edge{7, 11, 641});
    edges.push_back(edge{7, 12, 631});

    edges.push_back(edge{8, 9, 337});
    edges.push_back(edge{8, 11, 861});

    edges.push_back(edge{9, 10, 678});
    edges.push_back(edge{9, 11, 967});

    edges.push_back(edge{10, 11, 1024});

	edges.push_back(edge{11, 12, 1013});


	
	// edges.push_back(edge{4,5,4});
	// edges.push_back(edge{4,11,8});
	// edges.push_back(edge{5,6,8});
	// edges.push_back(edge{5,11,11});
	// edges.push_back(edge{6,7,7});
	// edges.push_back(edge{6,12,2});
	// edges.push_back(edge{6,9,4});
	// edges.push_back(edge{7,8,9});
	// edges.push_back(edge{7,9,14});
	// edges.push_back(edge{8,9,10});
	// edges.push_back(edge{9,10,2});
	// edges.push_back(edge{10,11,1});
	// edges.push_back(edge{10,12,6});
	// edges.push_back(edge{11,12,7});

	// construct graph
	Graph graph(edges, ARRAY_SIZE);
	load_kernelArrays(graph);

	// print adjacency list representation of graph
	printGraph(graph);

	//Source vertex as first non empty vertex in adjacency List
	//Or modify this to take from input file
	int source;
	for(int i = 0; i<nonEmptyIndices.size(); i++) {
			if (nonEmptyIndices[i]) {
				source = i;
			break;
		}
	}	
	
	//printf("source:%d\n", source);
	
  	//printf("Before Prim\n");

	hipEvent_t start, stop;
    	hipEventCreate(&start);
        hipEventCreate(&stop);

        //printf("Running global reduce\n");
    	hipEventRecord(start, 0);

	primMST(graph, ARRAY_SIZE, source);

	 hipEventRecord(stop, 0);

        hipEventSynchronize(stop);
    	float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Parallel Elpased Time in ms:%f\n", elapsedTime);

	
  	//printf("After Prim\n");

	//printf("T size:%d\n", T_index);
	//printf("MST in iterator\n");
	printf("\n====================================\n");
	printf("Minimum Spanning Tree using Prim \n");
	for (int i =0; i<T_index; i++) {
		fromTo e = T[i]; 
		printf("%d - %d\n", e.from, e.to); 
	}

	
	return 0; 
} 

//Reference: https://www.geeksforgeeks.org/prims-mst-for-adjacency-list-representation-greedy-algo-6/
// https://www.techiedelight.com/graph-implementation-using-stl/

